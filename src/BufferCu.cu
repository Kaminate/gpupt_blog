#include "BufferCu.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>
#include <>

#include <iostream>
#include <vector>


namespace gpupt
{

bufferCu::bufferCu(int Size, void *InitData) : Size(Size){
    hipMalloc((void**)&this->Data, Size);
    if(InitData != nullptr) hipMemcpy(this->Data, InitData, Size, hipMemcpyHostToDevice);
}

// Destructor
bufferCu::~bufferCu() {
    Destroy();
}

void bufferCu::Destroy()
{
    hipFree(Data); 
}

// Update SSBO data
void bufferCu::updateData(const void* data, size_t dataSize) {
    hipMemcpy(this->Data, data, dataSize, hipMemcpyHostToDevice);
}

void bufferCu::updateData(size_t offset, const void* data, size_t dataSize) {
    hipMemcpy((void*)((uint8_t*)this->Data + offset), data, dataSize, hipMemcpyHostToDevice);
}

}
