#include "hip/hip_runtime.h"
#pragma once
#include "BVH.h"
#include <glm/glm.hpp>
using namespace glm;
using namespace gpupt;

#define PI_F 3.141592653589
#define INVALID_ID -1


#define MATERIAL_TYPE_MATTE 0
#define MATERIAL_TYPE_PBR   1

__device__ u32 Width;
__device__ u32 Height;
__device__ triangle *TriangleBuffer;
__device__ triangleExtraData *TriangleExBuffer;
__device__ bvhNode *BVHBuffer;
__device__ u32 *IndicesBuffer;
__device__ indexData *IndexDataBuffer;
__device__ bvhInstance *TLASInstancesBuffer;
__device__ tlasNode *TLASNodes;
__device__ camera *Cameras;
__device__ tracingParameters *Parameters;
__device__ material *Materials;
__device__ hipTextureObject_t SceneTextures;
__device__ int LightsCount;
__device__ light *Lights;


#define MAIN() \
__global__ void TraceKernel(glm::vec4 *RenderImage, int _Width, int _Height, \
                            triangle *_AllTriangles, triangleExtraData *_AllTrianglesEx, bvhNode *_AllBVHNodes, u32 *_AllTriangleIndices, indexData *_IndexData, bvhInstance *_Instances, tlasNode *_TLASNodes,\
                            camera *_Cameras, tracingParameters* _TracingParams, material *_Materials, hipTextureObject_t _SceneTextures, lights *_Lights)

#define INIT() \
    Width = _Width; \
    Height = _Height; \
    TriangleBuffer = _AllTriangles; \
    TriangleExBuffer = _AllTrianglesEx; \
    BVHBuffer = _AllBVHNodes; \
    IndicesBuffer = _AllTriangleIndices; \
    IndexDataBuffer = _IndexData; \
    TLASInstancesBuffer = _Instances; \
    TLASNodes = _TLASNodes; \
    Cameras = _Cameras; \
    Parameters = _TracingParams; \
    Materials = _Materials; \
    SceneTextures = _SceneTextures; \
    LightsCount = _Lights->LightsCount; \
    Lights = _Lights->Lights; \


#define IMAGE_SIZE(Img) \
    ivec2(Width, Height)

#define GLOBAL_ID() \
    uvec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y)

#define FN_DECL __device__

#define INOUT(Type) Type &

#define GET_ATTR(Obj, Attr) \
    Obj->Attr


__device__ void imageStore(vec4 *Image, ivec2 p, vec4 Colour)
{
    Image[p.y * Width + p.x] = Colour;
}

__device__ vec4 imageLoad(vec4 *Image, ivec2 p)
{
    return Image[p.y * Width + p.x];
}

__device__ vec4 textureSample(hipTextureObject_t _SceneTextures, glm::vec3 Coords)
{
    static int NumLayersX = 8192 / 512;
    int LayerInx = Coords.z;
    
    int LocalCoordX = Coords.x * 512;
    int LocalCoordY = Coords.y * 512;

    int XOffset = (LayerInx % NumLayersX) * 512;
    int YOffset = (LayerInx / NumLayersX) * 512;

    int CoordX = XOffset + LocalCoordX;
    int CoordY = YOffset + LocalCoordY;

    uchar4 TexValue = tex2D<uchar4>(_SceneTextures, CoordX, CoordY);
    vec4 TexValueF = vec4((float)TexValue.x / 255.0f, (float)TexValue.y / 255.0f, (float)TexValue.z / 255.0f, (float)TexValue.w / 255.0f);
    return TexValueF;
}

 
#include "../../resources/PathTraceCode.cpp"


__device__ float ToSRGB(float Col) {
  return (Col <= 0.0031308f) ? 12.92f * Col
                             : (1 + 0.055f) * pow(Col, 1 / 2.4f) - 0.055f;
}

__device__ glm::vec3 ToSRGB(glm::vec3 Col)
{
    return glm::vec3(
        ToSRGB(Col.x),
        ToSRGB(Col.y),
        ToSRGB(Col.z)
    );
}

__global__ void TonemapKernel(glm::vec4 *Input,glm::vec4 *Output, int Width, int Height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < Width && y < Height) {    
        int index = y * Width + x;

        glm::vec3 Col = ToSRGB(Input[y * Width + x]);
        Output[y * Width + x] = vec4(Col, 1);    
    }
}